#include <iostream>
#include <vector>
#include "hip/hip_runtime.h"

using namespace std;

typedef vector<int> Vi;

const int S = 50, S2 = S * S;
const int M = 100;
const bool VERIFY = false;

// https://oeis.org/A337663
//       n = 1,  2,  3,  4,  5,  6,    7,    8,    9, ..
// A337663 = 1, 16, 28, 38, 49, 60, >=67, >=74, >=81, ..
//
// (mathologer, https://www.youtube.com/watch?v=m4Uth-EaTZ8)
// ("Some lower bounds for 7 <= n <= 9.", https://oeis.org/A337663/a337663_1.txt)

// n           1 2 3      4      5        6          7
// init-boards 1 5 137 8825 576451 38177587 2517580016 (distance <=4)

long long INIT_BOARDS[] = {
  // n
  // 1  2    3     4       5         6           7
  1, 1, 5, 137, 8825, 576451, 38177587, 2517580016L,
};
int PRINT_THRES[] = {
  // n
  // 1   2   3   4   5   6     7     8     9
  // A337663
  // 1, 16, 28, 38, 49, 60, >=67, >=74, >=81, ..
  0, 1, 16, 28, 38, 47, 57,   61,   64,   74,
};

struct board {
  int delta[8];

  uint16_t board[S2];
  uint16_t counts[S2];

  uint16_t candNext[S2 + M + 1];
  uint16_t candPrev[S2 + M + 1];
  uint16_t candList[S2];
  int candListPtr, candLevel;
  uint16_t candPtr[M + 1];

  int maxFound;
  uint16_t best[S2];
  long long searchedPositions;

  __device__
  void init();
  __device__
  void place(int pos, int value);
  __device__
  void unlink(int pos);
  __device__
  void link(int pos);

  void print(bool best = false);
  void printOnes();
  void verify();

  __device__
  bool search1();
};

int Pos(int row, int col) {
  return row * S + col;
}
int posRow(int pos) {
  return pos / S;
}
int posCol(int pos) {
  return pos % S;
}

__device__
void board::init() {
  delta[0] = -S - 1;
  delta[1] = -S;
  delta[2] = -S + 1;
  delta[3] = -1;
  delta[4] = 1;
  delta[5] = S - 1;
  delta[6] = S;
  delta[7] = S + 1;
  for (int i = 0; i < S2; i++) {
    board[i] = 0;
    counts[i] = 0;
    candPrev[i] = 0;
    candNext[i] = 0;
    candList[i] = 0;
    best[i] = 0;
  }
  for (int i = 0; i <= M; i++) {
    candPrev[S2 + i] = S2 + i;
    candNext[S2 + i] = S2 + i;
    candPtr[i] = 0;
  }
  candListPtr = 0;
  candLevel = 2;
  maxFound = 0;
  searchedPositions = 0;
}

__device__
void board::place(int pos, int value) {
  // cout << "place " << posRow(pos)-20 << ',' << posCol(pos)-20 << ' ' << value << endl;
  for (int i = 0; i < 8; i++) {
    int neigh = pos + delta[i];
    if (board[neigh] == 0) {
      unlink(neigh);
    }
  }
  if (board[pos] == 0) {
    unlink(pos);
  }
  board[pos] += value;
  if (board[pos] == 0) {
    link(pos);
  }
  for (int i = 0; i < 8; i++) {
    int neigh = pos + delta[i];
    counts[neigh] += value;
    if (board[neigh] == 0) {
      link(neigh);
    }
  }
  if (board[pos] > maxFound) {
    maxFound = board[pos];
    for (int i = 0; i < S2; i++) {
      best[i] = board[i];
    }
  }
}

__device__
void board::unlink(int pos) {
  int count = counts[pos];
  if (count > M) {
    return;
  }
  candNext[candPrev[pos]] = candNext[pos];
  candPrev[candNext[pos]] = candPrev[pos];
}

__device__
void board::link(int pos) {
  int count = counts[pos];
  if (count > M) {
    return;
  }
  candPrev[pos] = S2 + count;
  candNext[pos] = candNext[S2 + count];
  candPrev[candNext[pos]] = pos;
  candNext[S2 + count] = pos;
}

void board::print(bool best) {
  if (VERIFY && !best) {
    verify();
  }
  uint16_t *board = best ? this->best : this->board;
  int minRow = S, minCol = S, maxRow = 0, maxCol = 0;
  for (int row = 0; row < S; row++) {
    for (int col = 0; col < S; col++) {
      int pos = Pos(row, col);
      if (board[pos] > 0) {
        minRow = min(minRow, row);
        minCol = min(minCol, col);
        maxRow = max(maxRow, row);
        maxCol = max(maxCol, col);
      }
    }
  }
  for (int row = minRow; row <= maxRow; row++) {
    for (int col = minCol; col <= maxCol; col++) {
      int pos = Pos(row, col);
      if (board[pos] > 0) {
        cout.width(3);
        cout << board[pos];
      } else {
        cout << "  .";
      }
    }
    cout << endl;
  }
}

void board::printOnes() {
  if (VERIFY) {
    verify();
  }
  uint16_t *board = this->board;
  int minRow = S, minCol = S, maxRow = 0, maxCol = 0;
  int n = 0;
  for (int row = 0; row < S; row++) {
    for (int col = 0; col < S; col++) {
      int pos = Pos(row, col);
      if (board[pos] == 1) {
        minRow = min(minRow, row);
        minCol = min(minCol, col);
        maxRow = max(maxRow, row);
        maxCol = max(maxCol, col);
        n++;
      }
    }
  }
  cout << n;
  for (int row = minRow; row <= maxRow; row++) {
    for (int col = minCol; col <= maxCol; col++) {
      int pos = Pos(row, col);
      if (board[pos] == 1) {
        cout << ' ' << (row - minRow) << ' ' << (col - minCol);
      }
    }
  }
  cout << endl;
}

// verify() function, to check all counts, candidate lists, and placements on board!
void board::verify() {
  int minRow = S, minCol = S, maxRow = 0, maxCol = 0;
  for (int row = 0; row < S; row++) {
    for (int col = 0; col < S; col++) {
      int pos = Pos(row, col);
      if (board[pos] > 0) {
        minRow = min(minRow, row);
        minCol = min(minCol, col);
        maxRow = max(maxRow, row);
        maxCol = max(maxCol, col);
      }
    }
  }
  for (int row = minRow - 1; row <= maxRow + 1; row++) {
    for (int col = minCol - 1; col <= maxCol + 1; col++) {
      int pos = Pos(row, col);
      if (board[pos] == 0) {
        int count = 0;
        for (int i = 0; i < 8; i++) {
          count += board[pos + delta[i]];
        }
        if (counts[pos] != count) {
          print();
          printf("count mismatch %d != %d at %d,%d\n", counts[pos], count, row - minRow, col - minCol);
          throw "count mismatch";
        }
        if (count >= 2 && count <= M) {
          bool found = false;
          for (int candPos = candNext[S2 + count]; candPos < S2; candPos = candNext[candPos]) {
            if (candPos == pos) {
              found = true;
              break;
            }
          }
          if (!found) {
            print();
            printf("count not found among candidates for %d at %d,%d\n", count, row - minRow, col - minCol);
            for (int candPos = candNext[S2 + count]; candPos < S2; candPos = candNext[candPos]) {
              printf(" cand %d,%d", posRow(candPos) - minRow, posCol(candPos) - minCol);
            }
            printf("\n");
            throw "candidate missing mismatch";
          }
        }
      }
    }
  }
  for (int count = 2; count <= M; count++) {
    for (int pos = candNext[S2 + count]; pos != S2 + count; pos = candNext[pos]) {
      if (board[pos] != 0) {
        print();
        printf("placed %d in candidates for %d at %d,%d", board[pos], count, posRow(pos) - minRow, posCol(pos) - minCol);
        for (int candPos = candNext[S2 + count]; candPos < S2; candPos = candNext[candPos]) {
          printf(" cand %d,%d", posRow(candPos) - minRow, posCol(candPos) - minCol);
        }
        printf("\n");
        throw "candidate already placed mismatch";
      }
      if (board[pos] == 0 && counts[pos] != count && count >= 2) {
        print();
        printf("wrong count %d among candidates for %d at %d,%d", counts[pos], count, posRow(pos) - minRow, posCol(pos) - minCol);
        for (int candPos = candNext[S2 + count]; candPos < S2; candPos = candNext[candPos]) {
          printf(" cand %d,%d", posRow(candPos) - minRow, posCol(candPos) - minCol);
        }
        printf("\n");
        throw "candidate wrong count mismatch";
      }
    }
  }
}

__device__
bool board::search1() {
  //if (VERIFY) {
  //  verify();
  //}
  //while (candLevel >= 2) {
  candList[candListPtr++] = S2;
  for (int pos = candNext[S2 + candLevel]; pos < S2; pos = candNext[pos]) {
    candList[candListPtr++] = pos;
  }
  candPtr[candLevel] = candListPtr;

  int pos = candList[--candPtr[candLevel]];
  while (pos >= S2) {
    candLevel--;
    if (candLevel < 2) {
      return false;
    }
    pos = candList[candPtr[candLevel]];
    place(pos, -candLevel);
    candListPtr = candPtr[candLevel];
    pos = candList[--candPtr[candLevel]];
  }
  place(pos, candLevel);
  searchedPositions++;
  candLevel++;
  //}
  return true;
}

long long hist[M];
long long histSearched[M];
long long maxSearched[M];
void print_hist(int maxFound) {
  cout << maxFound << " hist";
  for (int i = 2; i <= maxFound; i++) {
    cout << ' ' << hist[i];
  }
  cout << endl;
  cout << maxFound << " avg";
  for (int i = 2; i <= maxFound; i++) {
    cout << ' ' << histSearched[i] / (hist[i] != 0 ? hist[i] : 1);
  }
  cout << endl;
  cout << maxFound << " max";
  for (int i = 2; i <= maxFound; i++) {
    cout << ' ' << maxSearched[i];
  }
  cout << endl;
}

time_t t0 = time(0);
int maxFound = 0;
long long initBoards = 0, searchedPositions = 0;
int n;

int read_boards(int max_boards, int *&boards, int *&results) {
  int board_n = 0;
  for (int board_i = 0; board_i < max_boards; board_i++) {
    if (!(cin >> n)) {
      break;
    }
    if (boards == NULL) {
      hipMallocManaged(&boards, n * max_boards * sizeof(int));
      hipMallocManaged(&results, 2 * max_boards * sizeof(int));
      for (int i = 0; i < n * max_boards; i++) {
        boards[i] = 0;
      }
      for (int i = 0; i < 2 * max_boards; i++) {
        results[i] = 0;
      }
    }
    for (int i = 0; i < n; i++) {
      int row, col;
      cin >> row >> col;
      int pos = Pos(row + 20, col + 20);
      boards[board_i * n + i] = pos;
    }
    results[board_i * 2] = 0;
    results[board_i * 2 + 1] = 0;
    board_n = board_i + 1;

    initBoards++;
    if (initBoards % 10000 == 0) {
      if (initBoards % 100000 == 0) {
        print_hist(maxFound);
      }
      double runSeconds = time(0) - t0;
      double totalHours = runSeconds * INIT_BOARDS[n] / initBoards / 3600;
      double etaHours = totalHours - runSeconds / 3600;
      printf("%d ", maxFound);
      printf("%.0fs tot %.1fh eta %.1fh ", runSeconds, totalHours, etaHours);
      printf("initial boards: %lld/%lld searched positions: %lld\n",
          initBoards, INIT_BOARDS[n], searchedPositions);
    }
  }
  return board_n;
}

__global__
void stepping_main(int board_n, int n, int *boards, int *results) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  board b;
  int board_i = index;
  bool new_board = true;
  while (board_i < board_n) {
    if (new_board) {
      b.init();
      for (int i = 0; i < n; i++) {
        b.place(boards[board_i * n + i], 1);
      }
    }
    new_board = false;
    if (!b.search1()) {
      results[board_i * 2] = b.maxFound;
      results[board_i * 2 + 1] = b.searchedPositions;
      new_board = true;
      board_i += stride;
    }
  }
}

void collect_results(int board_n, int *boards, int *results) {
  for (int board_i = 0; board_i < board_n; board_i++) {
    int biMaxFound = results[board_i * 2];
    int biSearchedPositions = results[board_i * 2 + 1];
    hist[biMaxFound]++;
    histSearched[biMaxFound] += biSearchedPositions;
    if (biSearchedPositions > maxSearched[biMaxFound]) {
      maxSearched[biMaxFound] = biSearchedPositions;
    }
    searchedPositions += biSearchedPositions;
    if (biMaxFound > maxFound) {
      maxFound = biMaxFound;
    }
    if (biMaxFound >= PRINT_THRES[n]) {
      cout << biMaxFound << " max " << n;
      for (int i = 0; i < n; i++) {
        int pos = boards[board_i * n + i];
        int row = posRow(pos) - 20, col = posCol(pos) - 20;
        cout << ' ' << row << ' ' << col;
      }
      cout << endl;
      // TODO: capture best board!
    }
  }
}

void stepping_end() {
  {
    print_hist(maxFound);
    double runSeconds = time(0) - t0;
    double totalHours = runSeconds / 3600;
    printf("%d ", maxFound);
    printf("%.0fs tot %.1fh ", runSeconds, totalHours);
    printf("initial boards: %lld/%lld searched positions: %lld\n",
        initBoards, INIT_BOARDS[n], searchedPositions);
  }
}

#ifndef STEPPING_THREAD
int main() {
  int *boards = 0, *results = 0;
  while (true) {
    const int AT_A_TIME = 1024 * 256 + 176 * 256; //10240;
    int board_n = read_boards(AT_A_TIME, boards, results);
    if (board_n == 0) {
      break;
    }

printf("pre-sync1\n");
    hipDeviceSynchronize();
printf("post-sync1\n");
    int blockSize = 256;
    int numBlocks = (board_n + blockSize - 1) / blockSize;
    if (numBlocks > 16) {
      numBlocks = 16;
    }
    //stepping_main<<<1, 1>>>(board_n, n, boards, results);
    stepping_main<<<numBlocks, blockSize>>>(board_n, n, boards, results);
printf("pre-sync2\n");
    hipDeviceSynchronize();
printf("post-sync2\n");

    printf("boards %p results %p results[0] %d,%d,%d\n", boards, results, results[0], results[1], results[2]);
    collect_results(board_n, boards, results);
  }
  stepping_end();
  //cudaProfilerStop();
  //cuProfilerStop();
  return 0;
}
#endif
